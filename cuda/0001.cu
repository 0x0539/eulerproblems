#include <algorithm>
#include <iostream>

#include <hip/hip_runtime.h>


__global__ void run(int* perThreadSums, size_t numItemsPerThread, size_t numItems) {
  int sum = 0;
  int t = blockIdx.x * blockDim.x + threadIdx.x; 
  int i = t * numItemsPerThread;
  int j = i + numItemsPerThread;
  if (numItems < j) {
    j = (int) numItems;
  }
  for (int k = i; k < j; k++) {
    sum += k * ((k % 3 == 0) || (k % 5 == 0));
  }
  perThreadSums[t] = sum;
}

int main() {
  size_t numItems = 1000;
  size_t numThreadsPerBlock = 128;
  size_t numItemsPerThread = 8;
  size_t numItemsPerBlock = numThreadsPerBlock * numItemsPerThread;
  size_t numBlocks = numItems / numItemsPerBlock;
  if (numItems % numItemsPerBlock != 0) {
    numBlocks++;
  }
  size_t numThreads = numThreadsPerBlock * numBlocks;
  
  int* perThreadSumsDevice;
  int perThreadSumsHost[numThreads];
  hipMalloc((void **)&perThreadSumsDevice, sizeof(int)*numThreads);

  run<<<numBlocks, numThreadsPerBlock>>>(perThreadSumsDevice, numItemsPerThread, numItems);

  hipMemcpy(perThreadSumsHost, perThreadSumsDevice, sizeof(int)*numThreads, hipMemcpyDeviceToHost);
  int sum = 0;
  for (int i = 0; i < numThreads; i++) {
    sum += perThreadSumsHost[i];
  }
  std::cout << sum << std::endl;
  hipFree(perThreadSumsDevice);
}
