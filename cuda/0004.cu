#include <algorithm>
#include <iostream>
#include <hip/hip_runtime.h>


__device__ int get_digits(uint32_t* x, uint32_t n) {
  int d = 0;
  while (n > 0) {
    x[d] = n % 10;
    n /= 10;
    d++;
  }
  return d;
}

__global__ void run(uint32_t* perBlockResult) {
  __shared__ uint32_t maxPalindrome;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    maxPalindrome = 0;
  }
  __syncthreads();
  uint32_t n = x * y;
  uint32_t digits[6] = {0};
  uint32_t d = get_digits(digits, n);
  bool palindrome = true;
  for (int i = 0; i < d / 2; i++) {
    int r = i;
    int l = d - 1 - i;
    palindrome = palindrome && (digits[r] == digits[l]);
  }
  if (palindrome && x < 1000 && y < 1000) {
    atomicMax(&maxPalindrome, n);
  }
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    perBlockResult[blockIdx.x * gridDim.y + blockIdx.y] = maxPalindrome;
  }
}


int main() {
  dim3 blockDim = dim3(16, 16);
  dim3 gridDim = dim3(64, 64);  // 64 * 16 = 1024 (>999)

  size_t numBlocks = sizeof(uint32_t) * gridDim.x * gridDim.y;

  uint32_t* perBlockResultDevice;
  hipMalloc((void **)&perBlockResultDevice, sizeof(uint32_t) * numBlocks);

  std::cout << "running with numBlocks " << numBlocks << std::endl;

  run<<<gridDim, blockDim>>>(perBlockResultDevice);

  std::cout << "done" << std::endl;

  uint32_t* perBlockResultHost = (uint32_t*)malloc(sizeof(uint32_t) * numBlocks);
  hipMemcpy(perBlockResultHost, perBlockResultDevice, sizeof(uint32_t) * numBlocks, hipMemcpyDeviceToHost);
  
  uint32_t maxPalindrome = 0;
  for (int i = 0; i < gridDim.x; i++) {
    for (int j = 0; j < gridDim.y; j++) {
      uint32_t p = perBlockResultHost[i * gridDim.y + j];
      if (p > maxPalindrome) {
        maxPalindrome = p;
      }
    }
  }

  std::cout << maxPalindrome << std::endl;

  hipFree(perBlockResultDevice);
  free(perBlockResultHost);
}
